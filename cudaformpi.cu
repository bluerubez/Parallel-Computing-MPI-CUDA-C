
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4
#define threads_per_block 4

__global__ void simpleKernel(float *out, float *in)
{
   int index;
   index = blockIdx.x*blockDim.x+threadIdx.x;
   if(index<N)
   {
   out[index]=in[index]*in[index]*in[index];
   }
}

extern "C"

void GPU_STUFF(int device)
{
hipSetDevice(device);
printf("Device number %d \n",device);

float *s_host, *r_host;
int pad, i;
float *r_device;
float *s_device;
size_t size;
pad = threads_per_block - (N % threads_per_block);
size = (N+pad)*sizeof(float);
s_host = (float *)malloc(size);
r_host = (float *)malloc(size);
hipMalloc(&s_device, size);
hipMalloc(&r_device, size);

dim3 threads(threads_per_block);
dim3 grid( (N+pad)/threads_per_block );

for (i=0;i<N;i++)
{
r_host[i]=i;
}

hipMemcpy(r_device, r_host, size, hipMemcpyHostToDevice);

simpleKernel <<< grid, threads >>> (s_device,r_device);

hipMemcpy(s_host, s_device, size, hipMemcpyDeviceToHost);

for (i=0;i<N;i++)
{
printf("%3f %3f\n",r_host[i],s_host[i]);
}

free(s_host);
free(r_host);
hipFree(r_device);
hipFree(s_device);

}






