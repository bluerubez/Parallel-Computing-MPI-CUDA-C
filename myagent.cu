//George Lees Jr.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 1000
#include <iostream>
#include <ctime>

extern "C"


__global__ void NearestNeighborKernel(float* A_x_d, float* A_y_d ,float* B_x_d, float* B_y_d, float* A_agentstrength_d, float* B_agentstrength_d)
{
int j_min_d;
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx < N)//so threads don't go past the bounds
	{
	
	float dist_min = 3.40282e38f;
	
	for(int j = 0; j < N; j++)
		{
		if(j == idx) continue;

		float dist_vec_d = sqrt(pow((A_x_d[idx]-B_x_d[j]),2)+pow((A_y_d[idx]-B_y_d[j]),2));

		//syncthreads

		if(dist_vec_d < dist_min)
			{
			dist_min = dist_vec_d;
			j_min_d= j;
			}

		//syncthreads

		if(dist_min<0.02 && B_agentstrength_d[j_min_d]>0.001)
    		{
        	/* Simple fight model - weakest agent killed */

    		if(A_agentstrength_d[j]<B_agentstrength_d[j_min_d])
        	{
        	A_agentstrength_d[j]=0;
        	}
    		else
        	{
        	B_agentstrength_d[j_min_d]=0;
        	}
    		}
	}
	}
}

int main(void)
{

// Declare variables and pointers on host to send to gpu
float *A_x, *A_y, *B_x, *B_y;
float *A_agentstrength;
float *B_agentstrength;
int vector_size;
vector_size = N*sizeof(float);
A_x = (float *)malloc(vector_size);
A_y = (float *)malloc(vector_size);
B_x = (float *)malloc(vector_size);
B_y = (float *)malloc(vector_size);
A_agentstrength = (float *)malloc(vector_size);
B_agentstrength = (float *)malloc(vector_size);


int i, time;
float delx, dely;
int total_dead_A[N];
int total_dead_B[N];
FILE *myfile;
FILE *myfile2;

// Declare variables on device
float *A_x_d;
float *A_y_d;
float *B_x_d;
float *B_y_d;
float *A_agentstrength_d;
float *B_agentstrength_d;
hipMalloc(&A_x_d, vector_size);
hipMalloc(&A_y_d, vector_size);
hipMalloc(&B_x_d, vector_size);
hipMalloc(&B_y_d, vector_size);
hipMalloc(&A_agentstrength_d, vector_size);
hipMalloc(&B_agentstrength_d, vector_size);


/* open files to write results */

myfile = fopen("agentresults.dat","w");
myfile2 = fopen("agentsummary.dat","w");

/* A loop to set the initial positions and strengths of 200 agents
 * (100 in team A and 100 in team B) which are random distributed
 * in a region with corners (-4,-4),(-4,4), (4,-4) and (4,4).
 * Strengths are assigned as random numbers between 0.5 and 1.0.
 * If agentstrength is less than 0.001 it is considered dead.        */

for (i=1;i<=N;++i)
{
A_x[i] = -4.0 + 8.0*((float) rand() / (float) (RAND_MAX-1));
A_y[i] = -4.0 + 8.0*((float) rand() / (float) (RAND_MAX-1));
B_x[i] = -4.0 + 8.0*((float) rand() / (float) (RAND_MAX-1));
B_y[i] = -4.0 + 8.0*((float) rand() / (float) (RAND_MAX-1));

A_agentstrength[i]=0.5 + 0.5* ((float) rand() / (float) (RAND_MAX-1));
B_agentstrength[i]=0.5 + 0.5* ((float) rand() / (float) (RAND_MAX-1));
}

// Start the time loop

for(time=1;time<=100;++time)
{

/* Loop through all agents on both teams (A and B)
 * and update their positions by adding random numbers to their
 * position (x,y). Note that if they move outside a region bounded
 * by the corners (-5,-5), (-5,5), (5,-5) and (5,5) they are put
 * back in the region.
 */

	for (i=1;i<=N;++i)
	{

	    if(A_agentstrength[i]>0.001)
	    {

		    delx = (float) rand() / (float) (RAND_MAX-1) - 0.5 ;
		    dely = (float) rand() / (float) (RAND_MAX-1) - 0.5;

	    	A_x[i] = A_x[i] + delx;
		A_y[i] = A_y[i] + dely;

		/* If, by adding the movement we move the agent out of the confining region
		 * then move the agent back to stay in region   */

		if(A_x[i]>5) {A_x[i]=A_x[i]-delx;}
		if(A_x[i]<-5) {A_x[i]=A_x[i]+delx;}
		if(A_y[i]>5) {A_y[i]=A_y[i]-dely;}
		if(A_y[i]<-5) {A_y[i]=A_y[i]+dely;}

	    }
	    if(B_agentstrength[i]>0.001)
	    {

		    delx = (float) rand() / (float) (RAND_MAX-1) - 0.5;
		    dely = (float) rand() / (float) (RAND_MAX-1) - 0.5;

	    	B_x[i] = B_x[i] + delx;
		B_y[i] = B_y[i] + dely;

		/* If, by adding the movement we move the agent out of the confining region
		 * then move the agent back to stay in region   */

		if(B_x[i]>5) {B_x[i]=B_x[i]-delx;}
		if(B_x[i]<-5) {B_x[i]=B_x[i]+delx;}
		if(B_y[i]>5) {B_y[i]=B_y[i]-dely;}
		if(B_y[i]<-5) {B_y[i]=B_y[i]+dely;}

	    }
	}

	/* Loop through all agents on both teams A and for each agent
 	* find the nearest individual on team B. Then fight with it.
 	* Loser dies.*/
	//&&
	/* If the nearest agent on team B is close enough to the agent i on team A
	* and not already dead then fight */
	// transfer vectors A_x, A_y, B_x, B_y to gpu
	hipMemcpy(A_x_d, A_x, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(A_y_d, A_y, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_x_d, B_x, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_y_d, B_y, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(A_agentstrength_d, A_agentstrength, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_agentstrength_d, B_agentstrength, vector_size, hipMemcpyHostToDevice);
	

	NearestNeighborKernel<<< 4, 512>>> (A_x_d, A_y_d, B_x_d,
	B_y_d, A_agentstrength_d, B_agentstrength_d);

	hipMemcpy(A_x, A_x_d, vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(A_y, A_y_d, vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(B_x, B_x_d, vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(B_y, B_y_d, vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(A_agentstrength,A_agentstrength_d, vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(B_agentstrength,B_agentstrength_d, vector_size, hipMemcpyDeviceToHost);

   /* Print out all agent positions and strengths for both teams */

    for(i=1;i<=N;++i)
    {
    fprintf(myfile, "%f %f %f %f %f %f \n",A_x[i],A_y[i],A_agentstrength[i],B_x[i],B_y[i],B_agentstrength[i]);
    }


    /* Compute number dead for both teams */

     total_dead_A[time]=0;
     total_dead_B[time]=0;
     for(i=1;i<=N;++i)
     {
     if(A_agentstrength[i]<0.001){total_dead_A[time]=total_dead_A[time]+1;}
     if(B_agentstrength[i]<0.001){total_dead_B[time]=total_dead_B[time]+1;}
     }
     fprintf(myfile2, "%d %d  \n",total_dead_A[time],total_dead_B[time]);


/* Go back for another time step */

}

free(A_x);free(A_y);free(B_x);free(B_y);
hipFree(A_x_d);
hipFree(A_y_d);
hipFree(B_x_d);
hipFree(B_y_d);

    /* Close files */

    if(myfile!=NULL)
    {
    fclose(myfile);
    }
    if(myfile2!=NULL)
    {
    fclose(myfile2);
    }


    return EXIT_SUCCESS;
}
